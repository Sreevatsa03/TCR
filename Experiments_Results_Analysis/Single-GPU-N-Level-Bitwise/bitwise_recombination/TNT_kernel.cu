#include "hip/hip_runtime.h"
#ifndef _TNT_KERNEL_H_
#define _TNT_KERNEL_H_

#include <stdio.h>

//DB information
__constant__ int const_numDB1 = 169;				//total number of DB1 chewbacks
__constant__ char const_d_DB1[425];				//constant memory allocation for DB chewbacks minus full chew back
//__constant__ char const_d_DB1[1448];				//constant memory allocation for DB chewbacks minus full chew back
__constant__ int const_d_DB1_base[169];				//constant memory contains location of each starting sequence in d_DB1 
__constant__ int const_d_numOccurrenceDB1[169];		//Number of ways for particular DB1 chewback 
__constant__ int const_d_numUniqueCharDB1[169];		//number of characters in a unique occurence of DB1
//V information
__constant__ int const_numV = 342;					//total number of V sequences in all V files
__constant__ char const_d_V[913];					//holds all V chewback sequences
//__constant__ char const_d_V[3107];					//holds all V chewback sequences
__constant__ int const_d_V_base[342];				//contains the starting index of each V sequence
__constant__ int const_d_numUniqueCharV[342];		//number of characters in a unique occurence of V
//J information
__constant__ int const_numJ = 271;					//total number of J sequences in all J files
__constant__ char const_d_J[908];					//holds all J chewback sequences
//__constant__ char const_d_J[3210];					//holds all J chewback sequences
__constant__ int const_d_J_base[271];				//contains the starting index of each J sequence
__constant__ int const_d_numUniqueCharJ[271];		//number of characters in a unique occurence of J


__constant__ int c_DB_Full_Chew_Occur;				//current V sequence
__constant__ int c_Vnum;							//current V sequence
__constant__ int c_Dnum;							//current D sequence
__constant__ int c_Jnum;							//current J sequence
__constant__ int c_n;								//current n value

__constant__ int c_V_Begin;							//Base index for V sequences
__constant__ int c_V_End;							//End index for V sequences
__constant__ int c_J_Begin;							//Base index for J sequences
__constant__ int c_J_End;							//End index for J sequences

__constant__ int const_d_VJ_Pairs[NUM_V_FILES*NUM_J_FILES];
__constant__ int const_VJ_Pair_Base[NUM_V_FILES*NUM_J_FILES];
__constant__ int c_NUM_V_FILES = 20;
__constant__ int c_NUM_J_FILES = 12;

/////////////////////////////////////////////////
//kernel for 64 threads or less
/////////////////////////////////////////////////
__global__ void
TNT_kernel_InVivo64(unsigned int* d_Results, char* d_InVivo_cp64)	
{

	volatile __shared__ char iterSeq_sm[16]; //the thread block size we will use for this kernel is 64
	volatile __shared__ int result_sm[128];  //the max thread-block size

	//The four possible bases
//	char base[4] = {'A', 'T', 'G', 'C'};
	unsigned char base[4] = { 0x00, 0x01, 0x02, 0x03 };	
	__shared__ int arraylocal[9*128];//{Vnum,Jnum, sum, num_Seqs,gl_index, pairBase, seqLen, g_tid, length};
//	char nSeq[12];					//will hold a single n combination
	 arraylocal[0*128 + threadIdx.x]= c_Vnum;				//current V file --Vnum 
	 arraylocal[1*128 + threadIdx.x] = c_Jnum;				//current J file --Jnum
	int sh_index;					//used as a shared memory index
	 //sum;						//holds an iterative sum for result
	
	//obtain a unique global index for each thread in the grid
	arraylocal[7*128 + threadIdx.x] = blockIdx.x * blockDim.x + threadIdx.x; // unsigned int  g_tid
/*
	if(c_n){
		nSeq[0]  = base[g_tid%4];					//n = 1
		nSeq[1]  = base[(g_tid+(g_tid/4))%4];		//n = 2
		nSeq[2]  = base[(g_tid+(g_tid/16))%4];		//n = 3
		nSeq[3]  = base[(g_tid+(g_tid/64))%4];		//n = 4
		nSeq[4]  = base[(g_tid+(g_tid/256))%4];		//n = 5
		nSeq[5]  = base[(g_tid+(g_tid/1024))%4];	//n = 6
		nSeq[6]  = base[(g_tid+(g_tid/4096))%4];	//n = 7
		nSeq[7]  = base[(g_tid+(g_tid/16384))%4];	//n = 8
		nSeq[8]  = base[(g_tid+(g_tid/65536))%4];	//n = 9
		nSeq[9]  = base[(g_tid+(g_tid/262144))%4];	//n = 10
		nSeq[10] = base[(g_tid+(g_tid/1048576))%4]; //n = 11
		nSeq[11] = base[(g_tid+(g_tid/4194304))%4];	//n = 12
	}
*/

	unsigned char encNSeq[3];
	if(c_n){
		encNSeq[0] = ( base[ arraylocal[7*128 + threadIdx.x] % 4 ] << 6 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 4 )) % 4 ] << 4 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 16 )) % 4 ] << 2 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 64 )) % 4] );
		encNSeq[1] = ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] /   256 )) % 4 ] << 6 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] /   1024 )) % 4 ] << 4 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] /    4096 )) % 4 ] << 2 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] /   16384 )) % 4 ] );
		encNSeq[2] = ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 65536 )) % 4 ] << 6 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 262144 )) % 4 ] << 4 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 1048576 )) % 4 ] << 2 ) | ( base[( arraylocal[7*128 + threadIdx.x] + ( arraylocal[7*128 + threadIdx.x] / 4194304 )) % 4 ] );
	}	

	//get the number of InVivo VJ sequences we need to go through
	arraylocal[3*128 + threadIdx.x] = const_d_VJ_Pairs[arraylocal[0*128 + threadIdx.x]*12 + arraylocal[1*128 + threadIdx.x]];				//multiply by 12. Number of J files. --num_Seqs

	//int whichSeq; //which sequence is our current thread-block working on in the scope of current VJ
	//int seqLen;	  //length of our current sequence
	arraylocal[5*128 + threadIdx.x] = const_VJ_Pair_Base[arraylocal[0*128 + threadIdx.x]*12 + arraylocal[1*128 + threadIdx.x]] * 16; //The base address for a given VJ pair --pairBase

	//iterate through all InVivo combinations for current VJ pair
	for( int i = 0; i < arraylocal[3*128 + threadIdx.x]; i++ ) {

		result_sm[ threadIdx.x ] = 0;
		arraylocal[2*128 + threadIdx.x] = 0; //reset our result

		__syncthreads();

		//store an InVivo combination into the shared memory "iterResults_sm[]"
		if( blockDim.x < 16 ) {										//iter through VJ seq if block dim < 64. There's only 1 block
			for( int j = 0; j < ( 16 / blockDim.x ); j++ ) {		//iterations = sequence allocation / block size
				//int k = j * blockDim.x + threadIdx.x;				//create local SM index
				arraylocal[4*128 + threadIdx.x] = ( arraylocal[5*128 + threadIdx.x]  + i * 16 ) + j * blockDim.x + threadIdx.x;			//create global memory index
				iterSeq_sm[ j * blockDim.x + threadIdx.x ] = d_InVivo_cp64[ arraylocal[4*128 + threadIdx.x] ];    	//read the current InVivo sequence from the global memory
			}
		}
		else{	//only threads < 16 will read inVivo data
			if( threadIdx.x < 16 ) {
				arraylocal[4*128 + threadIdx.x] = ( arraylocal[5*128 + threadIdx.x]  + i * 16 ) + threadIdx.x;		//create global memory index
				iterSeq_sm[ threadIdx.x ] = d_InVivo_cp64[ arraylocal[4*128 + threadIdx.x] ];		//read the current InVivo sequence from the global memory
			}
		}

		//if(blockDim.x > 1) 
		__syncthreads();

		//get the length of current sequence for all threads in current thread-block
		// iterSeq_sm[0] = [7:3] V, [2:0] MS-3bits J
		// iterSeq_sm[1] = [7] LS-1bit J, [6:0] Length of InVivo Sequence
		// iterSeq_sm[2] to iterSeq_sm[15] = InVivo code

		unsigned char tempChar0, tempChar1, tempChar2;

		char getChar;
		getChar = iterSeq_sm[1] & 0x7F;				// Mask [7]th bit, as Length = [6:0] ( 0b01111111 = 0x7F )
		arraylocal[6*128 + threadIdx.x] = (int)getChar;						// seqLen = Length of InVivo sequence --seqLen

//		printf("i: %d, seqLen: %d, iterSeq_sm: %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d\n", i, seqLen, iterSeq_sm[0], iterSeq_sm[1], iterSeq_sm[2], iterSeq_sm[3], iterSeq_sm[4], iterSeq_sm[5], iterSeq_sm[6], iterSeq_sm[7], iterSeq_sm[8], iterSeq_sm[9], iterSeq_sm[10], iterSeq_sm[11], iterSeq_sm[12], iterSeq_sm[13], iterSeq_sm[14], iterSeq_sm[15]);
//		printf("c_V_Begin: %d, c_V_End: %d \n", c_V_Begin, c_V_End );

		//set our shared memory index to the base of the sequence characters in shared array
		int byteCount = 0;
		int binCount = 0;
		int accuBinCount = 0;
		int k;
		
		int tempAccuBinCount = 0;
		int tempShIndex = 0;

		//int length;				// Length of each generated sequence
		bool Vmatch = true;		// Is there a V sequence match?
		bool seqMatch = true;		// Is the entire sequence a match?

		//////////////////////////////////////////////////////////////////////////////////
		//First compare our InVivo Sequences containing 'V' = Vn, VnJ, VnDn, VnDnJ
		//////////////////////////////////////////////////////////////////////////////////

		// Get all the V sequences:
		for ( int Vindx = c_V_Begin; Vindx < c_V_End; Vindx++ ) {
			
			// Initialize values:
			Vmatch = true;
			seqMatch = true;

			/////////////////////////////////////////////////////////
			//Compare InVivo Sequence to Vn comb with D and J chewed 
			/////////////////////////////////////////////////////////

			arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharV[ Vindx ] + c_n; //length
			if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] ) {
//				printf("Length matches: i: %d, Vindx: %d, c_n: %d, Vlen: %d, length: %d, seqLen: %d\n", i, Vindx, c_n, const_d_numUniqueCharV[ Vindx ], length, seqLen );
				
				sh_index = 2;													// InVivo sequence starts from iterSeq_sm[2]
				accuBinCount = 0;

				byteCount = (( const_d_numUniqueCharV[ Vindx ] ) / 4 );			// Calculates the full bytes of V
				binCount = ( const_d_numUniqueCharV[ Vindx ] % 4 ) * 2;			// Calculates the overflow bits of V
				k = const_d_V_base[ Vindx ];									// Starting address of V sequence

				// Compare the full bytes of V with InVivo sequence
				for ( int m = 0; m < byteCount; m++ ) {
					tempChar0 = const_d_V[ k ];
					tempChar1 = iterSeq_sm[ sh_index ];
					if ( tempChar0 != tempChar1 ) {
						Vmatch = false;
						break;
					}
					sh_index++;
					k++;
				}

				// V does not match => get next V sequence => go to line 158 ( V for loop )
				if ( Vmatch == false ) {
					continue;
				}

				// Compare the overflow bits of V with Invivo sequence
				tempChar0 = ((( const_d_V[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
				tempChar1 = ((( iterSeq_sm[ sh_index ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
				if ( tempChar0 != tempChar1 ) {
					Vmatch = false;
					continue;
				}

				// If we've reached this point: V sequence matches !!
				accuBinCount += binCount;
				byteCount = ( c_n / 4 );
				binCount = (( c_n % 4 ) * 2 );

				// Compare full bytes of n with InVivo sequence
				
				if ( c_n != 0 ) {
					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = encNSeq[ m ];
						tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
						if ( tempChar0 != tempChar1 ) {
							seqMatch = false;
							break;
						}
						sh_index++;
					}

					// Compare the overflow bits of V with Invivo sequence
					tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
					tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar2 ) {
						seqMatch = false;
					}
				}

				if ( seqMatch == true ) {
					// If we've reached this point: the sequence matches
					arraylocal[2*128 + threadIdx.x] += c_DB_Full_Chew_Occur;
//					printf("Vn: i: %d, nval: %d, pairBase: %d, Vindx: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, valueAdd: %d \n", i, c_n, (pairBase / 16), Vindx, length, seqLen, g_tid, sum, c_DB_Full_Chew_Occur);
				}
			}

			if ( Vmatch == false ) {
				continue;
			}


			/////////////////////////////////////////////////////////
			//Compare InVivo Sequence to VnJ comb with D chewed
			/////////////////////////////////////////////////////////

			for ( int Jindx = c_J_Begin; Jindx < c_J_End; Jindx++ ) {
				
				arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharV[ Vindx ] + const_d_numUniqueCharJ[ Jindx ] + c_n;
				if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] ) {
//					printf(" VnJ Length matches: i: %d, Vindx: %d, Jindx: %d, c_n: %d, Vlen: %d, Jlen: %d, length: %d, seqLen: %d\n", i, Vindx, Jindx, c_n, const_d_numUniqueCharV[ Vindx ], const_d_numUniqueCharJ[ Jindx ], length, seqLen );
					seqMatch = true;
					sh_index = 2;
					accuBinCount = 0;
					byteCount = (( const_d_numUniqueCharV[ Vindx ] ) / 4 );		// Calculates the full bytes of V
					binCount = ( const_d_numUniqueCharV[ Vindx ] % 4 ) * 2;		// Calculates the overflow bits of V
					k = const_d_V_base[Vindx];									// Starting address of V sequence

					// Compare the full bytes of V with InVivo sequence
					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = const_d_V[ k ];
						tempChar1 = iterSeq_sm[ sh_index ];
						if ( tempChar0 != tempChar1 ) {
							Vmatch = false;
							break;
						}
						sh_index++;
						k++;
					}
					
					// V does not match => get next V sequence => break J loop 
					if ( Vmatch == false ) {
						break;
					}
					
					// Compare the overflow bits of V with Invivo sequence
					tempChar0 = ((( const_d_V[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ((( iterSeq_sm[ sh_index ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar1 ) {
						Vmatch = false;
						break;
					}

					// If we've reached this point: V sequence matches !!
					accuBinCount += binCount;
					byteCount = ( c_n / 4 );
					binCount = (( c_n % 4 ) * 2 );

					if ( c_n != 0 ) {
						// Compare the full bytes of n with InVivo sequence
						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = encNSeq[ m ];
							tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));	
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							sh_index++;
						}
						
						if ( seqMatch == false ) {
							break;
						}

						// Compare the overflow bits of n with Invivo sequence
						tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							break;
						}
					}

					accuBinCount += binCount;
					if ( accuBinCount >= 8 ) {
						sh_index++;
					}
					accuBinCount %= 8;

					byteCount = (( const_d_numUniqueCharJ[ Jindx ] ) / 4 );			// Calculates the full bytes of J
					binCount = ( const_d_numUniqueCharJ[ Jindx ] % 4 ) * 2;			// Calculates the overflow bits of J
					k = const_d_J_base[ Jindx ];									// Starting address of V sequence

					// Compare the full bytes of J with InVivo sequence
					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = const_d_J[ k ];
						tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
						if ( tempChar0 != tempChar1 ) {
							seqMatch = false;
							break;
						}
						sh_index++;
						k++;
					}
					
					if ( seqMatch == false ) {
						continue;
					}

					// Compare the overflow bits of J with Invivo sequence
					tempChar0 = ((( const_d_J[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
					tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar2 ) {
						seqMatch = false;
						continue;
					}
					
					if ( seqMatch == true ) {
						arraylocal[2*128 + threadIdx.x] += c_DB_Full_Chew_Occur; 		// The sequence matches
//						printf("VnJ: i: %d, n: %d, pairBase: %d, Vindx: %d, Jindx: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, c_n, (pairBase / 16), Vindx, Jindx, length, seqLen, g_tid, sum, c_DB_Full_Chew_Occur);
					}
				}
			}

			if ( Vmatch == false ) {
				continue;
			}

			/////////////////////////////////////////////////////////
			//Compare InVivo Sequence to VnDn comb with J chewed
			/////////////////////////////////////////////////////////

			for ( int Dindx = 0; Dindx < const_numDB1; Dindx++ ) {
				
				seqMatch = true;
				arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharV[ Vindx ] + const_d_numUniqueCharDB1[ Dindx ] + c_n;
				if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] ) {
//					printf(" VnJ Length matches: i: %d, Vindx: %d, Dindx: %d, c_n: %d, Vlen: %d, Dlen: %d, length: %d, seqLen: %d\n", i, Vindx, Dindx, c_n, const_d_numUniqueCharV[ Vindx ], const_d_numUniqueCharDB1[ Dindx ], length, seqLen );
					seqMatch = true;
					sh_index = 2;
					accuBinCount = 0;
					byteCount = (( const_d_numUniqueCharV[ Vindx ] ) / 4 );		// Calculates the full bytes of V
					binCount = ( const_d_numUniqueCharV[ Vindx ] % 4 ) * 2;		// Calculates the overflow bits of V
					k = const_d_V_base[Vindx];									// Starting address of V sequence

					// Compare the full bytes of V with InVivo sequence
					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = const_d_V[ k ];
						tempChar1 = iterSeq_sm[ sh_index ];
						if ( tempChar0 != tempChar1 ) {
							Vmatch = false;
							break;
						}
						sh_index++;
						k++;
					}
					
					// V does not match => get next V sequence => break J loop 
					if ( Vmatch == false ) {
						break;
					}
					
					// Compare the overflow bits of V with Invivo sequence
					tempChar0 = ((( const_d_V[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ((( iterSeq_sm[ sh_index ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar1 ) {
						Vmatch = false;
						break;
					}

//					if ( i == 157 ) {
//						printf("VnD Byte match: i: %d, pairBase: %d, c_n: %d, Vindx: %d, Dindx: %d, Vlen: %d, Dlen: %d, length: %d, seqLen: %d, g_tid: %d\n", i, (pairBase / 16), c_n, Vindx, Dindx, const_d_numUniqueCharV[ Vindx ], const_d_numUniqueCharDB1[ Dindx ], length, seqLen, g_tid);
//					}

					accuBinCount += binCount;
					
					for ( int nlen = 0; nlen < ( c_n + 1 ); nlen++ ) {

						seqMatch = true;
						tempAccuBinCount = accuBinCount;
						tempShIndex = sh_index;
						byteCount = ( nlen / 4 );
						binCount = (( nlen % 4 ) * 2 );

						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = encNSeq[ m ];
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));	
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
						}

						if ( seqMatch == false ) {
							continue;
						}
						
						// Compare the overflow bits of n with Invivo sequence
						tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							continue;
						}
						
						tempAccuBinCount += binCount;
						if ( tempAccuBinCount >= 8 ) {
							tempShIndex++;
						}
						tempAccuBinCount %= 8;

						byteCount = ( const_d_numUniqueCharDB1[ Dindx ] / 4 );
						binCount = (( const_d_numUniqueCharDB1[ Dindx ] % 4 ) * 2 );
						k = const_d_DB1_base[ Dindx ];									// Starting address of V sequence

						// Compare the full bytes of D with InVivo sequence
						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = const_d_DB1[ k ];
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
							k++;
						}

						if ( seqMatch == false ) {
							continue;
						}
						
						// Compare the overflow bits of D with Invivo sequence
						tempChar0 = ((( const_d_DB1[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							continue;
						}

						tempAccuBinCount += binCount;
						if ( tempAccuBinCount >= 8 ) {
							tempShIndex++;
						}
						tempAccuBinCount %= 8;

//						printf("VnD Byte match: i: %d, nlen: %d, c_n: %d, Vindx: %d, Dindx: %d, g_tid: %d\n", i, nlen, c_n, Vindx, Dindx, g_tid );
//						Worked till here!

						byteCount = ( nlen / 4 );
						binCount = (( nlen % 4 ) * 2 );

						for ( int m = 0; m < (( c_n - nlen ) / 4 ); m++ ) {
							tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
							byteCount++;
						}

						if ( seqMatch == false ) {
							continue;
						}

						tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
//						if ( i == 157 ) {
//							printf("i: %d, nlen: %d, c_n: %d, Vindx: %d, Dindx: %d, g_tid: %d \n\t byteCount: %d, binCount: %d, tempAccuBinCount: %d, tempShIndex: %d \n\t encNSeq: %d, tempChar0: %d \n\t iterSeq_sm: %d %d, tempChar1: %d, tempChar2: %d \n", i, nlen, c_n, Vindx, Dindx, g_tid, byteCount, binCount, tempAccuBinCount, tempShIndex, encNSeq[ byteCount ], tempChar0, iterSeq_sm[ tempShIndex ], iterSeq_sm[ tempShIndex + 1 ], tempChar1, tempChar2);
//						}
						binCount = ((( c_n - nlen ) % 4 ) * 2 );
						tempChar0 = ((( tempChar0 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar1 ) {
							seqMatch = false;
							continue;
						}

						if ( seqMatch == true )	{
							arraylocal[2*128 + threadIdx.x] += const_d_numOccurrenceDB1[ Dindx ];
//							printf("VnDn: i: %d,pairBase: %d, Vindx: %d, Dindx: %d, nlen: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, (pairBase / 16), Vindx, Dindx, nlen, length, seqLen, g_tid, sum, const_d_numOccurrenceDB1[Dindx]);
						}
					}

				}

				/////////////////////////////////////////////////////////
				//Compare InVivo Sequence to VnDnJ comb with no chewback
				/////////////////////////////////////////////////////////

				for ( int Jindx = c_J_Begin; Jindx < c_J_End; Jindx++ ) {	

					seqMatch = true;
					arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharV[Vindx] + const_d_numUniqueCharJ[Jindx] + const_d_numUniqueCharDB1[Dindx] + c_n;
					if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] )
					{	
						sh_index = 2;													// InVivo sequence starts from iterSeq_sm[2]
						accuBinCount = 0;
						byteCount = (( const_d_numUniqueCharV[ Vindx ] ) / 4 );			// Calculates the full bytes of V
						binCount = ( const_d_numUniqueCharV[ Vindx ] % 4 ) * 2;			// Calculates the overflow bits of V
						k = const_d_V_base[Vindx];										// Starting address of V sequence

						// Compare the full bytes of V with InVivo sequence
						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = const_d_V[ k ];
							tempChar1 = iterSeq_sm[ sh_index ];
							if ( tempChar0 != tempChar1 ) {
								Vmatch = false;
								break;
							}
							sh_index++;
							k++;
						}
						
						// V does not match => get next V sequence => break J loop 
						if ( Vmatch == false ) {
							break;
						}
						
						// Compare the overflow bits of V with Invivo sequence
						tempChar0 = ((( const_d_V[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ((( iterSeq_sm[ sh_index ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar1 ) {
							Vmatch = false;
							break;
						}
						
//						printf("VnDnJ_Vmatch: i: %d, nval: %d, pairBase: %d, Vindx: %d, Dindx: %d, Jindx: %d, seqLen: %d, Vlen: %d, Dlen: %d, Jlen: %d, g_tid: %d \n", i, c_n, (pairBase / 16), Vindx, Dindx, Jindx, seqLen, const_d_numUniqueCharV[Vindx], const_d_numUniqueCharDB1[Dindx], const_d_numUniqueCharJ[Jindx], g_tid);

						accuBinCount += binCount;
						
						for ( int nlen = 0; nlen < ( c_n + 1 ); nlen++ ) {

							seqMatch = true;
							tempAccuBinCount = accuBinCount;
							tempShIndex = sh_index;
							byteCount = ( nlen / 4 );
							binCount = (( nlen % 4 ) * 2 );

							for ( int m = 0; m < byteCount; m++ ) {
								tempChar0 = encNSeq[ m ];
								tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));	
								if ( tempChar1 != tempChar0 ) {
									seqMatch = false;
									break;
								}
								tempShIndex++;
							}	

							if ( seqMatch == false ) {
								continue;
							}
						
							// Compare the overflow bits of n with Invivo sequence
							tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							if ( tempChar0 != tempChar2 ) {
								seqMatch = false;
								continue;
							}
						
							tempAccuBinCount += binCount;
							if ( tempAccuBinCount >= 8 ) {
								tempShIndex++;
							}
							tempAccuBinCount %= 8;

							byteCount = ( const_d_numUniqueCharDB1[ Dindx ] / 4 );
							binCount = (( const_d_numUniqueCharDB1[ Dindx ] % 4 ) * 2 );
							k = const_d_DB1_base[ Dindx ];									// Starting address of V sequences

							// Compare the full bytes of D with InVivo sequence
							for ( int m = 0; m < byteCount; m++ ) {
								tempChar0 = const_d_DB1[ k ];
								tempChar2 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
								if ( tempChar0 != tempChar2 ) {
									seqMatch = false;
									break;
								}
								tempShIndex++;
								k++;
							}

							if ( seqMatch == false ) {
								continue;
							}
						
							// Compare the overflow bits of D with Invivo sequence
							tempChar0 = ((( const_d_DB1[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							if ( tempChar0 != tempChar2 ) {
								seqMatch = false;
								continue;
							}

							tempAccuBinCount += binCount;
							if ( tempAccuBinCount >= 8 ) {
								tempShIndex++;
							}
							tempAccuBinCount %= 8;

							byteCount = ( nlen / 4 );
							binCount = (( nlen % 4 ) * 2 );	

							for ( int m = 0; m < (( c_n - nlen ) / 4 ); m++ ) {
								tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
								tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
								if ( tempChar0 != tempChar1 ) {
									seqMatch = false;
									break;
								}
								tempShIndex++;
								byteCount++;
							}

							if ( seqMatch == false ) {
								continue;
							}

							tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							
							binCount = ((( c_n - nlen ) % 4 ) * 2 );
							tempChar0 = ((( tempChar0 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								continue;
							}

							tempAccuBinCount += binCount;
							if ( tempAccuBinCount >= 8 ) {
								tempShIndex++;
							}
							tempAccuBinCount %= 8;

							byteCount = (( const_d_numUniqueCharJ[ Jindx ] ) / 4 );			// Calculates the full bytes of J
							binCount = ( const_d_numUniqueCharJ[ Jindx ] % 4 ) * 2;			// Calculates the overflow bits of J
							k = const_d_J_base[ Jindx ];									// Starting address of V sequence

							for ( int m = 0; m < byteCount; m++ ) {
								tempChar0 = const_d_J[ k ];
								tempChar2 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
								if ( tempChar0 != tempChar2 ) {
									seqMatch = false;
									break;
								}
								tempShIndex++;
								k++;
							}

							if ( seqMatch == false ) {
								break;
							}

							tempChar0 = ((( const_d_J[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							if ( tempChar0 != tempChar2 ) {
								seqMatch = false;
								break;
							}

							if ( seqMatch == true ) {
								arraylocal[2*128 + threadIdx.x] += const_d_numOccurrenceDB1[ Dindx ];
//								printf("VnDnJ: i: %d, n: %d, pairBase: %d, Vindx: %d, Jindx: %d, Dindx: %d, nlen: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, c_n, (pairBase / 16), Vindx, Jindx, Dindx, nlen, length, seqLen, g_tid, sum, const_d_numOccurrenceDB1[Dindx]); 
							}
						}
					}
				}

				if ( Vmatch == false ) {
					break;
				}

			}

			if ( Vmatch == false ) {
				continue;
			}
		}

		//////////////////////////////////////////////////////////////////////////////////
		//Compare our InVivo Sequences containing 'nDn' = nDn, nDnJ
		//////////////////////////////////////////////////////////////////////////////////

		// Get all the D sequences:
		for ( int Dindx = 0; Dindx < const_numDB1; Dindx++ ) {

			arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharDB1[Dindx] + c_n;
			if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] )
			{	

				//int tempAccuBinCount = 0;
				//int tempShIndex = 0;

				for ( int nlen = 0; nlen < ( c_n + 1 ); nlen++ ) {

					seqMatch = true;
					tempAccuBinCount = 0;
					tempShIndex = 2;
					byteCount = ( nlen / 4 );
					binCount = (( nlen % 4 ) * 2 );

					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = encNSeq[ m ];
						tempChar2 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));	
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							break;
						}
						tempShIndex++;
					}	

					if ( seqMatch == false ) {
						continue;
					}
						
					// Compare the overflow bits of n with Invivo sequence
					tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
					tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar2 ) {
						seqMatch = false;
						continue;
					}
						
					tempAccuBinCount += binCount;
					if ( tempAccuBinCount >= 8 ) {
						tempShIndex++;
					}
					tempAccuBinCount %= 8;

					byteCount = ( const_d_numUniqueCharDB1[ Dindx ] / 4 );
					binCount = (( const_d_numUniqueCharDB1[ Dindx ] % 4 ) * 2 );
					k = const_d_DB1_base[ Dindx ];									// Starting address of D sequences

					// Compare the full bytes of D with InVivo sequence
					for ( int m = 0; m < byteCount; m++ ) {
						tempChar0 = const_d_DB1[ k ];
						tempChar2 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							break;
						}
						tempShIndex++;
						k++;
					}

					if ( seqMatch == false ) {
						continue;
					}
						
					// Compare the overflow bits of D with Invivo sequence
					tempChar0 = ((( const_d_DB1[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
					tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar2 ) {
						seqMatch = false;
						continue;
					}

					tempAccuBinCount += binCount;
					if ( tempAccuBinCount >= 8 ) {
						tempShIndex++;
					}
					tempAccuBinCount %= 8;

					byteCount = ( nlen / 4 );
					binCount = (( nlen % 4 ) * 2 );	

					for ( int m = 0; m < (( c_n - nlen ) / 4 ); m++ ) {
						tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						if ( tempChar0 != tempChar1 ) {
							seqMatch = false;
							break;
						}
						tempShIndex++;
						byteCount++;
					}

					if ( seqMatch == false ) {
						continue;
					}

					tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							
					binCount = ((( c_n - nlen ) % 4 ) * 2 );
					tempChar0 = ((( tempChar0 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					tempChar1 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
					if ( tempChar0 != tempChar1 ) {
						seqMatch = false;
						continue;
					}

					if ( seqMatch == true ) {
						arraylocal[2*128 + threadIdx.x] += const_d_numOccurrenceDB1[ Dindx ];
//						printf("nDn: i: %d, n: %d, pairBase: %d, Dindx: %d, nlen: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, c_n, (pairBase / 16), Dindx, nlen, length, seqLen, g_tid, sum, const_d_numOccurrenceDB1[ Dindx ]);
					}
				}
			}

			/////////////////////////////////////////////////////////
			//Compare InVivo Sequence to nDnJ comb with V chewed
			/////////////////////////////////////////////////////////

			for ( int Jindx = c_J_Begin; Jindx < c_J_End; Jindx++ ) {
//				printf("nDnJ: i: %d, nval: %d, pairBase: %d,Dindx: %d, Jindx: %d, seqLen: %d, Dlen: %d, Jlen: %d, g_tid: %d \n", i, c_n, (pairBase / 16), Dindx, Jindx, seqLen, const_d_numUniqueCharDB1[Dindx], const_d_numUniqueCharJ[Jindx], g_tid);
				arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharJ[Jindx] + const_d_numUniqueCharDB1[Dindx] + c_n;
				if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] )
				{	

					//int tempAccuBinCount = 0;
					//int tempShIndex = 0;

					for ( int nlen = 0; nlen < ( c_n + 1 ); nlen++ ) {

						seqMatch = true;
						tempAccuBinCount = 0;
						tempShIndex = 2;
						byteCount = ( nlen / 4 );
						binCount = (( nlen % 4 ) * 2 );

						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = encNSeq[ m ];
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));	
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
						}	

						if ( seqMatch == false ) {
							continue;
						}
					
						// Compare the overflow bits of n with Invivo sequence
						tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							continue;
						}
						
						tempAccuBinCount += binCount;
						if ( tempAccuBinCount >= 8 ) {
							tempShIndex++;
						}
						tempAccuBinCount %= 8;

						byteCount = ( const_d_numUniqueCharDB1[ Dindx ] / 4 );
						binCount = (( const_d_numUniqueCharDB1[ Dindx ] % 4 ) * 2 );
						k = const_d_DB1_base[ Dindx ];									// Starting address of V sequences

						// Compare the full bytes of D with InVivo sequence
						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = const_d_DB1[ k ];
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
							k++;
						}

						if ( seqMatch == false ) {
							continue;
						}
						
						// Compare the overflow bits of D with Invivo sequence
						tempChar0 = ((( const_d_DB1[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							continue;
						}

						tempAccuBinCount += binCount;
						if ( tempAccuBinCount >= 8 ) {
							tempShIndex++;
						}
						tempAccuBinCount %= 8;

						byteCount = ( nlen / 4 );
						binCount = (( nlen % 4 ) * 2 );	

						for ( int m = 0; m < (( c_n - nlen ) / 4 ); m++ ) {
							tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
							tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							if ( tempChar0 != tempChar1 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
							byteCount++;
						}

						if ( seqMatch == false ) {
							continue;
						}

						tempChar0 = ( encNSeq[ byteCount ] << binCount ) | ((( encNSeq[ byteCount + 1 ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
								
						binCount = ((( c_n - nlen ) % 4 ) * 2 );
						tempChar0 = ((( tempChar0 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						tempChar1 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar1 ) {
							seqMatch = false;
							continue;
						}

						tempAccuBinCount += binCount;
						if ( tempAccuBinCount >= 8 ) {
							tempShIndex++;
						}
						tempAccuBinCount %= 8;

						byteCount = (( const_d_numUniqueCharJ[ Jindx ] ) / 4 );			// Calculates the full bytes of J
						binCount = ( const_d_numUniqueCharJ[ Jindx ] % 4 ) * 2;		// Calculates the overflow bits of J
						k = const_d_J_base[ Jindx ];									// Starting address of V sequence

						for ( int m = 0; m < byteCount; m++ ) {
							tempChar0 = const_d_J[ k ];
							tempChar2 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | ((( iterSeq_sm[ tempShIndex + 1 ] >> 2 ) & 0x3F ) >> ( 6 - tempAccuBinCount ));
							if ( tempChar0 != tempChar2 ) {
								seqMatch = false;
								break;
							}
							tempShIndex++;
							k++;
						}

						if ( seqMatch == false ) {
							break;
						}

						tempChar0 = ((( const_d_J[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount )) ;
						tempChar1 = ( iterSeq_sm[ tempShIndex ] << tempAccuBinCount ) | (( iterSeq_sm[ tempShIndex + 1] >> 2 ) & 0x3F >> ( 6 - tempAccuBinCount )); 
						tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
						if ( tempChar0 != tempChar2 ) {
							seqMatch = false;
							break;
						}

						if ( seqMatch == true ) {
							arraylocal[2*128 + threadIdx.x] += const_d_numOccurrenceDB1[ Dindx ];
//							printf("nDnJ: i: %d, n: %d, pairBase: %d, Jindx: %d, Dindx: %d, nlen: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, c_n, (pairBase / 16), Jindx, Dindx, nlen, length, seqLen, g_tid, sum, const_d_numOccurrenceDB1[ Dindx ]);
						}
					}
				}
			}
		}

		/////////////////////////////////////////////////////////
		//Compare InVivo Sequence to n comb with V, D and J chewed
		/////////////////////////////////////////////////////////

//		printf("n: i: %d, nval: %d, pairBase: %d, seqLen: %d, g_tid: %d \n", i, c_n, (pairBase / 16), seqLen, g_tid);
		if ( c_n == arraylocal[6*128 + threadIdx.x] )
		{	
			sh_index = 2;
			accuBinCount = 0;

			byteCount = ( c_n / 4 );
			binCount = (( c_n % 4 ) * 2 );

			// Compare full bytes of n with InVivo sequence
			
			for ( int m = 0; m < byteCount; m++ ) {
				tempChar0 = encNSeq[ m ];
				tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));	
				if ( tempChar1 != tempChar0 ) {
					seqMatch = false;
					break;
				}
				sh_index++;
			}

			if ( seqMatch == false ) {
				break;
			}

			// Compare the overflow bits of V with Invivo sequence
			tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
			tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
			tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
			if ( tempChar0 != tempChar2 ) {
				seqMatch = false;
			}

			if ( seqMatch == true ) {
				arraylocal[2*128 + threadIdx.x] += c_DB_Full_Chew_Occur;
//				printf("n: i: %d, pairBase: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, (pairBase / 16), length, seqLen, g_tid, sum, c_DB_Full_Chew_Occur);
			}
		}

		/////////////////////////////////////////////////////////
		//Compare InVivo Sequence to nJ comb with V and D chewed
		/////////////////////////////////////////////////////////
		for ( int Jindx = c_J_Begin; Jindx < c_J_End; Jindx++ ) {

//			printf("nJ: i: %d, nval: %d, pairBase: %d, Jindx: %d, seqLen: %d, Jlen: %d, g_tid: %d \n", i, c_n, (pairBase / 16), Jindx, seqLen, const_d_numUniqueCharJ[Jindx], g_tid);
			arraylocal[8*128 + threadIdx.x] = const_d_numUniqueCharJ[Jindx] + c_n;
			if ( arraylocal[8*128 + threadIdx.x] == arraylocal[6*128 + threadIdx.x] )
			{	
				sh_index = 2;
				accuBinCount = 0;

				byteCount = ( c_n / 4 );
				binCount = (( c_n % 4 ) * 2 );

				// Compare full bytes of n with InVivo sequence
				for ( int m = 0; m < byteCount; m++ ) {
					tempChar0 = encNSeq[ m ];
					tempChar2 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));	
					if ( tempChar2 != tempChar0 ) {
						seqMatch = false;
						break;
					}
					sh_index++;
				}

				if ( seqMatch == false ) {
					break;
				}

				// Compare the overflow bits of V with Invivo sequence
				tempChar0 = ((( encNSeq[ byteCount ] >> 2 ) & 0x3F ) >> ( 6 - binCount ));
				tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
				tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
				if ( tempChar0 != tempChar2 ) {
					seqMatch = false;
					break;
				}

				accuBinCount += binCount;
				if ( accuBinCount >= 8 ) {
					sh_index++;
				}
				accuBinCount %= 8;

				byteCount = (( const_d_numUniqueCharJ[ Jindx ] ) / 4 );			// Calculates the full bytes of J
				binCount = ( const_d_numUniqueCharJ[ Jindx ] % 4 ) * 2;			// Calculates the overflow bits of J
				k = const_d_J_base[ Jindx ];									// Starting address of V sequence

				// Compare the full bytes of J with InVivo sequence
				for ( int m = 0; m < byteCount; m++ ) {
					tempChar0 = const_d_J[ k ];
					tempChar2 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | ((( iterSeq_sm[ sh_index + 1 ] >> 2 ) & 0x3F ) >> ( 6 - accuBinCount ));
					if ( tempChar0 != tempChar2 ) {
						seqMatch = false;
						break;
					}
					sh_index++;
					k++;
				}

				if ( seqMatch == false ) {
					continue;
				}

				tempChar0 = ((( const_d_J[ k ] >> 2 ) & 0x3F ) >> ( 6 - binCount )) ;
				tempChar1 = ( iterSeq_sm[ sh_index ] << accuBinCount ) | (( iterSeq_sm[ sh_index + 1] >> 2 ) & 0x3F >> ( 6 - accuBinCount )); 
				tempChar2 = ((( tempChar1 >> 2 ) & 0x3F ) >> ( 6 - binCount ));
				if ( tempChar0 != tempChar2 ) {
					seqMatch = false;
					continue;
				}

				if ( seqMatch == true ) {
					arraylocal[2*128 + threadIdx.x] += c_DB_Full_Chew_Occur;
//					printf("nJ: i: %d, Jnum: %d, n: %d, pairBase: %d, Jindx: %d, length: %d, seqLen: %d, g_tid: %d, sum: %d, addVal: %d \n", i, Jnum, c_n, (pairBase / 16), Jindx, length, seqLen, g_tid, sum, c_DB_Full_Chew_Occur);
				}
			}
		}

		//-------------------------------------------------------------------------------------------------------
		//If only 1 thread-block, then we can write results to RAM using just InVivo sequence number
		//-------------------------------------------------------------------------------------------------------
		if( blockDim.x == 1 ) {		//if there is only 1 thread per block, just use i as global memory index. No need for reduction
			d_Results[ i ] = arraylocal[2*128 + threadIdx.x];
		}


		//-------------------------------------------------------------------------------------------------------
		//					Perform Reduction of Results if more than 1 thread-block				  
		//-------------------------------------------------------------------------------------------------------
		//reduction for current InVivo sequence in shared memory
		if( blockDim.x > 1 ) {
	
			result_sm[ threadIdx.x ] = arraylocal[2*128 + threadIdx.x];			//write a threads sum to the shared memory
			__syncthreads();						//make sure all sums have been written before proceeding

			int half = blockDim.x / 2;

			while( 1 ) {								//how many reductions we need
				if( threadIdx.x < half ) {				//only certain threads perform reduction
					result_sm[ threadIdx.x ] += result_sm[ threadIdx.x + half ];
				}
				__syncthreads();
				if( half == 1 ) break;
				half = half / 2;
			}

			__syncthreads();

			//write results to the global memory. Each thread-block writes 1 result for each InVivo Sequence i
			if( threadIdx.x == 0 ){					//we need only 1 thread in the thread block to write its result
				d_Results[ i * gridDim.x + blockIdx.x ] = result_sm[ 0 ];		//write our consolidated result into the global memory
			}

		}	//end result reduction
	}		//end iterating through InVivo Sequences
	
	return;
} 			//kernel done

#endif // #ifndef _TNT_KERNEL_H_


//	printf("Block Dim: %d, Block Idx: %d, Thread Idx: %d, g_tid: %d, encNSeq: %d,%d,%d\n", blockDim.x, blockIdx.x, threadIdx.x, g_tid, encNSeq[0], encNSeq[1], encNSeq[2]);

//	if ( g_tid == 0 ) {
//		printf("num_Seqs: %d, pairBase: %d\n", num_Seqs, pairBase);
//	}

//	if ( threadIdx.x == 1 ) {
//		printf("i: %d, threadIdx: %d, iterseq_sm: %d, gl_index: %d, d_InVivo_cp64: %d\n", i, threadIdx.x, iterSeq_sm[ threadIdx.x ], gl_index, d_InVivo_cp64[ gl_index ]);
//	}

//	if ( g_tid == 12 ) {
//		printf("i: %d, iterSeq_sm: %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d %d\n", i, iterSeq_sm[0], iterSeq_sm[1], iterSeq_sm[2], iterSeq_sm[3], iterSeq_sm[4], iterSeq_sm[5], iterSeq_sm[6], iterSeq_sm[7], iterSeq_sm[8], iterSeq_sm[9], iterSeq_sm[10], iterSeq_sm[11], iterSeq_sm[12], iterSeq_sm[13], iterSeq_sm[14], iterSeq_sm[15]);
//	}

//	__syncthreads();

//	if ( g_tid == 0 ) {
//		printf("i: %d, getChar: %d, seqLen: %d \n", i, getChar, seqLen);
//	}

//	if ( g_tid == 4 ) {
//		printf("getChar: %d, const_d_V[k]: %d \n", getChar, const_d_V[k]);
//	}

//	if ( g_tid == 4 ) {
//		printf("Byte V matches: i: %d, Vnum: %d, pairBase: %d, Vindx: %d, length: %d, seqLen: %d \n", i, Vnum, (pairBase / 16), Vindx, length, seqLen);
//	}

//	if ( g_tid == 4 ) {
//		printf("const_d_V: %d, iterSeq_sm: %d, V_tempChar0: %d, V_tempChar1: %d \n", const_d_V[k], iterSeq_sm[sh_index], tempChar0, tempChar1);
//	}

//	if ( g_tid == 4 ) {
//		printf("binCount V matches: i: %d, Vnum: %d, pairBase: %d, Vindx: %d, length: %d, seqLen: %d \n", i, Vnum, (pairBase / 16), Vindx, length, seqLen);
//	}

//	if ( seqMatch == true ) {
//		printf("Byte n matches: i: %d, Vnum: %d, pairBase: %d, Vindx: %d, length: %d, seqLen: %d, g_tid: %d \n", i, Vnum, (pairBase / 16), Vindx, length, seqLen, g_tid);
//	}

//	if ( seqMatch == true ) {
//		printf("n_tempChar0: %d, n_tempChar2: %d, g_tid: %d \n", tempChar0, tempChar2, g_tid);
//		printf("binCount n matches: i: %d, Vnum: %d, pairBase: %d, Vindx: %d, length: %d, seqLen: %d, g_tid: %d \n", i, Vnum, (pairBase / 16), Vindx, length, seqLen, g_tid);
//	}

