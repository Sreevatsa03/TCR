#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use tiling with shared memory for arbitrary size
  float CValue = 0;

  int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

  for (int i = 0; i < (TILE_WIDTH + numAColumns - 1)/TILE_WIDTH; i++) {

    if (i * TILE_WIDTH + threadIdx.x < numAColumns && row < numARows)
      As[threadIdx.y][threadIdx.x] = A[row * numAColumns + i*TILE_WIDTH + threadIdx.x];
    else
      As[threadIdx.y][threadIdx.x] = 0.0;

    if (i * TILE_WIDTH + threadIdx.y < numBRows && col < numBColumns)
      Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * numBColumns + col];
    else
      Bs[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();

    for (int j = 0; j < TILE_WIDTH; ++j)
      CValue += As[threadIdx.y][j] * Bs[j][threadIdx.x];

    __syncthreads();
  }

  if (row < numCRows && col < numCColumns)
    C[((blockIdx.y * blockDim.y + threadIdx.y) * numCColumns) + (blockIdx.x * blockDim.x) + threadIdx.x] = CValue;
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA; // A matrix on device
  float *deviceB; // B matrix on device
  float *deviceC; // C matrix on device
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C(you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
                            
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;   // set to correct value
  numCColumns = numBColumns;   // set to correct value
  //@@ Allocate the hostC matrix
  hostC = (float *) malloc(sizeof(float) * numCRows * numCColumns);
  
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **) &deviceA, sizeof(float)*numARows*numAColumns);
  hipMalloc((void **) &deviceB, sizeof(float)*numBRows*numBColumns);
  hipMalloc((void **) &deviceC, sizeof(float)*numCRows*numCColumns);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  // note that TILE_WIDTH is set to 16 on line number 13. 
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);    
  dim3 dimGrid((numCColumns/TILE_WIDTH) + 1, (numCRows/TILE_WIDTH) + 1, 1);
  
  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
